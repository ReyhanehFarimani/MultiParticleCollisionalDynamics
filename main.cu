#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <string>
#include <vector>
#include <random>
#include <exception>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "mpcd_macro.cuh"
#include "LEBC.cuh"
#include "reduction_sum.cuh"
#include "thermostat.cuh"
#include "streaming.cuh"
#include "collision.cuh"
#include "gallileain_inv.cuh"
#include "rerstart_file.cuh"
#include "gpu_md.cuh"
#include "md_analyser.cuh"
#include "begining.cuh"
#include "logging.cuh"

int main(int argc, const char* argv[])
{
    //Check for number of parsed argument:
    if( argc !=16 )
    {
        std::cout<<"Argument parsing failed!\n";
        std::string exeName = argv[0];
        std::cout<<exeName<<"\n";
        std::cout<<"Number of given arguments: "<<argc<<"\n";
        return 1;
    }

    // Setting the parsed argument:
    std::string inputfile= argv[1];         // Restart file name(The one reading from it!)
                                            // If the simulation start with t=0, put some 
                                            // dummy argument.
    std::string basename = argv[2];         // Output base name
    L[0] = atof(argv[3]);                   // Dimension of the simulation in x direction
    L[1] = atof(argv[4]);                   // Dimension of the simulation in y direction
    L[2] = atof(argv[5]);                   // Dimension of the simulation in z direction
    density = atoi(argv[6]);                // Density of the particles
    n_md = atoi(argv[7]);                   // Number of rings
    m_md = atof(argv[8]);                   // Number of monomer in each ring
    shear_rate = atof(argv[9]);             // Shear rate
    h_md = atof(argv[10]);                  // Md time step
    h_mpcd = atof(argv[11]);                // Mpcd time step
    swapsize = atoi(argv[12]);              // Output interval
    simuationtime = atoi(argv[13]);         // Final simulation step count
    TIME = atoi(argv[14]);                  // Starting time 
    topology = atoi(argv[15]);              // System topology 1 is a poly[n]catenane
                                            // 2 is the bonded ring.

    // Setting some constarint based on parsed argument  
    double ux =shear_rate * L[2];
    int Nc = L[0] * L[1] * L[2];            // Number of cells
    int N =density * Nc;                    // Number of MPCD particles
    int Nmd = n_md * m_md;                  // Number of MD particles

    // Setting the number of grid for parallel simulation,
    // It can be optimised based on GPU attributes, I did not care much about it!
    int grid_size = ((N + blockSize) / blockSize);
    
     // Random generator
     hiprandGenerator_t gen;
     hiprandCreateGenerator(&gen, 
         HIPRAND_RNG_PSEUDO_DEFAULT);
     // Setting seed for the simulation
     /* !NEVER EVER CHANGE THIS PART! */
     hiprandSetPseudoRandomGeneratorSeed(gen, 
         4294967296ULL^time(NULL));
     hiprandState *devStates;
     hipMalloc((void **)&devStates,
         blockSize * grid_size *sizeof(hiprandState));
     setup_kernel<<<grid_size, blockSize>>>(time(NULL), devStates);
    

    /*** Allocate device memory for mpcd particle ***/
    // position and velocity of MPCD particles:
    double *d_r_mpcd;
    hipMalloc((void**)&d_r_mpcd, sizeof(double) * N * 3);   

    double *d_v_mpcd;
    hipMalloc((void**)&d_v_mpcd, sizeof(double) * N * 3);

    // The mpcd index is used for sorting the particles,
    // into cell, more on collision module.
    int *d_index;
    hipMalloc((void**)&d_index, sizeof(int) * N);
    
    /*** Allocate device memory for box attributes ***/
    // d_L is a copy of box dimention in the GPU.
    // d_r is a ranfom vector for grid shifting.
    double *d_L, *d_r;   
    hipMalloc((void**)&d_L, sizeof(double) *3);
    hipMalloc((void**)&d_r, sizeof(double) *3); 
    
    // Allocate device memory for cells, 
    // d_u is for the mean momentum of cells
    double *d_v_cell;
    hipMalloc((void**)&d_v_cell, sizeof(double) * Nc * 3);
    
    // d_n is for saving number of MPCD particle in a cell
    // (important for the thermostat)
    // d_m is for saving the whole mass stored in each cell
    // (important for computing the mean momentum)
    int  *d_n, *d_m;
    hipMalloc((void**)&d_n, sizeof(int) * Nc);     
    hipMalloc((void**)&d_m, sizeof(int) * Nc);

    // The random angles attribated for each cell for Rotation step: 
    double *d_phi , *d_theta,*d_rot;
    hipMalloc((void**)&d_phi, sizeof(double) * Nc);    
    hipMalloc((void**)&d_theta, sizeof(double) * Nc);  

    // The rotation matrix of the cell:
    hipMalloc((void**)&d_rot, sizeof(double) * Nc * 9);

    // Allocate device memory for cell level thermostat atributes:
    double* d_e, *d_scalefactor;
    hipMalloc((void**)&d_e, sizeof(double) * Nc);             //kinetic energy of the cell particles.
    hipMalloc((void**)&d_scalefactor, sizeof(double) * Nc);    // scale factor to set the velocities distibuation
                                                                // to a desired gamma distribuation

    /*** Allocate device memory for md particle ***/
    /// postions, velocity, and acceleration:
    double *d_mdX, *d_mdY, *d_mdZ;
    hipMalloc((void**)&d_mdX, sizeof(double) * Nmd);    
    hipMalloc((void**)&d_mdY, sizeof(double) * Nmd);    
    hipMalloc((void**)&d_mdZ, sizeof(double) * Nmd);
    double *d_mdVx, *d_mdVy, *d_mdVz;
    hipMalloc((void**)&d_mdVx, sizeof(double) * Nmd);   
    hipMalloc((void**)&d_mdVy, sizeof(double) * Nmd);   
    hipMalloc((void**)&d_mdVz, sizeof(double) * Nmd);
    double *d_mdAx, *d_mdAy, *d_mdAz;
    hipMalloc((void**)&d_mdAx, sizeof(double) * Nmd);   
    hipMalloc((void**)&d_mdAy, sizeof(double) * Nmd);   
    hipMalloc((void**)&d_mdAz, sizeof(double) * Nmd);
    // This index will be used for sorting the MD particle in to the cells:
    int *d_mdIndex;
    hipMalloc((void**)&d_mdIndex, sizeof(int) * Nmd);

    // This attribute, is for matrix of polymer interaction in each direction
    double *md_Fx_holder , *md_Fy_holder , *md_Fz_holder;
    hipMalloc((void**)&md_Fx_holder, sizeof(double) * Nmd * Nmd);    
    hipMalloc((void**)&md_Fy_holder, sizeof(double) * Nmd * Nmd);    
    hipMalloc((void**)&md_Fz_holder, sizeof(double) * Nmd * Nmd);
    // These attributes can be changed and removed.
    // I know it is the worst way to compute force between particles,
    // But it is bugless and it does not affect my running speed much!
    // If you ever wanted to simulate a system with more MD particles,
    // You should modify this part!
    

    /* Simulation stars in this part! */
    if (TIME ==0)
    {
        start_simulation(basename, simuationtime, swapsize, d_L, d_mdX, d_mdY, d_mdZ,
                         d_mdVx, d_mdVy, d_mdVz, d_mdAx, d_mdAy, d_mdAz,
                         md_Fx_holder, md_Fy_holder, md_Fz_holder,
                         d_r_mpcd, d_v_mpcd, gen, grid_size);
    }
    // else 
    // {
    //     restarting_simulation(basename, inputfile, simuationtime, swapsize,
    //                          d_L, d_mdX, d_mdY, d_mdZ, d_mdVx, d_mdVy, d_mdVz,
    //                          d_mdAx, d_mdAy, d_mdAz, md_Fx_holder, md_Fy_holder, md_Fz_holder,
    //                          d_r_mpcd, d_v_mpcd, ux, N, Nmd, TIME, grid_size);
    // }

    /* Setting time for the simulation! */
    double real_time = TIME;                        // It is imprtant for us because of Lees Edwards PBC
    int T =simuationtime/swapsize +TIME/swapsize;   // Computing time for the loop based on logging frequency
    int delta = h_mpcd / h_md;                      // MD step calcilation based on MPCD time loop.  
    

    // Loop based on sampling frequncy"
    for (int t = TIME/swapsize; t<T; t++)
    {
        // Loop for calculation:
        for (int i =0; i<int(swapsize/h_mpcd); i++)
        {
            hiprandGenerateUniformDouble(gen, d_phi, Nc);
            hiprandGenerateUniformDouble(gen, d_theta, Nc);
            hiprandGenerateUniformDouble(gen, d_r, 3);

            // MPCD_streaming(d_x, d_y, d_z, d_vx, d_vy, d_vz, h_mpcd, N, grid_size);            

            // MD_streaming(d_mdX, d_mdY, d_mdZ, d_mdVx, d_mdVy, d_mdVz,
            //              d_mdAx , d_mdAy , d_mdAz ,md_Fx_holder, md_Fy_holder, md_Fz_holder,
            //              h_md , Nmd , density , d_L , ux , grid_size, delta,real_time);
            // // xyz_trj(basename + "_force.xyz", d_mdAx, d_mdAy, d_mdAz, Nmd);
            // Sort_begin(d_x, d_y, d_z, d_vx, d_index, d_mdX, d_mdY, d_mdZ,
            //             d_mdVx, d_mdIndex, ux, d_L, d_r, N, Nmd, real_time, grid_size);

            // MPCD_MD_collision(d_vx, d_vy, d_vz, d_index, d_mdVx, d_mdVy, d_mdVz,
            //                  d_mdIndex, d_ux, d_uy, d_uz, d_e, d_scalefactor, d_n, d_m,
            //                  d_rot, d_theta, d_phi, N, Nmd, Nc, devStates, grid_size);
            
            // Sort_finish(d_x, d_y, d_z,d_vx, d_index , 
            //              d_mdX, d_mdY, d_mdZ ,d_mdVx, d_mdIndex, ux, 
            //              d_L, d_r, N, Nmd, real_time, grid_size);
            
            real_time += h_mpcd;
                 

        }
        //logging:
        logging(basename + "_log.log", real_time, d_mdVx, d_mdVy, d_mdVz,
                 d_vx, d_vy, d_vz, N, Nmd, grid_size);
        xyz_trj(basename + "_traj.xyz", d_mdX, d_mdY, d_mdZ, Nmd);
        xyz_trj(basename + "_vel.xyz", d_mdVx, d_mdVy, d_mdVz, Nmd);
        xyz_trj(basename + "_force.xyz", d_mdAx, d_mdAy, d_mdAz, Nmd);
       
    }


    // // End of simualtion:
    // md_write_restart_file(basename, d_mdX , d_mdY , d_mdZ , d_mdVx , d_mdVy , d_mdVz , Nmd);
    // mpcd_write_restart_file(basename ,d_x , d_y , d_z , d_vx , d_vy , d_vz , N);

    // Free memory of the MPCD particles and cells:
    hipFree(d_r_mpcd);
    hipFree(d_v_mpcd); 
    hipFree(d_v_cell); 
    hipFree(d_rot); 
    hipFree(d_phi); 
    hipFree(d_theta);
    hipFree(devStates); 
    hipFree(d_e); 
    hipFree(d_scalefactor);
    // Free memory of the MD particles:
    hipFree(d_mdX);    
    hipFree(d_mdY);    
    hipFree(d_mdZ);
    hipFree(d_mdVx);   
    hipFree(d_mdVy);   
    hipFree(d_mdVz);
    hipFree(d_mdAx);   
    hipFree(d_mdAy);   
    hipFree(d_mdAz);
    hipFree(md_Fx_holder); 
    hipFree(md_Fy_holder); 
    hipFree(md_Fz_holder);
    hiprandDestroyGenerator(gen);

    std::cout<<"The program has terminated succesffuly at time:"<<real_time<<std::endl;
}
